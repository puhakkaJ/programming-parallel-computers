#include "hip/hip_runtime.h"
#include "cp.h"
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#define BASE_VALF 0.0


//straight copy from course material
static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

//straight copy from course material
static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

__global__ void mykernel(const float* data, float* result, int ny, int nx) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= ny || j >= ny)
        return;
    float v = BASE_VALF;
    for (int k = 0; k < nx; ++k) {
        float x = data[nx*j + k];
        float y = data[nx*i + k];
        float z = x * y;
        v += z;
    }
    result[ny*i + j] = v;
}

//straight copy from course material
void step(float* result, const float* d, int ny, int nx) {
    // Allocate memory & copy data to GPU
    float* dGPU = NULL;
    CHECK(hipMalloc((void**)&dGPU, ny * nx * sizeof(float)));
    float* rGPU = NULL; 
    CHECK(hipMalloc((void**)&rGPU, ny * ny * sizeof(float)));
    CHECK(hipMemcpy(dGPU, d, ny * nx * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(ny, dimBlock.x), divup(ny, dimBlock.y));
    mykernel<<<dimGrid, dimBlock>>>(dGPU, rGPU, ny, nx);
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, rGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}

/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
void correlate(int ny, int nx, const float *data, float *result) {
    float* data_norm = (float*) malloc(ny * nx * sizeof(float));

    for (int y = ny; y--;) {
        double sum = 0.0, mean = 0.0, pow_sum = 0.0, sqrt_sum_sqrt = 0.0;

        for (int x = nx; x--; ) {
            sum += data[x + y*nx];
        }

        mean = sum / nx;

        for (int x = nx; x--; ) {
            double normalized = data[x + y*nx] - mean;
            data_norm[x + y*nx] = normalized;
            pow_sum += normalized*normalized;
        }

        sqrt_sum_sqrt = sqrt(pow_sum);

        for (int x = nx; x--; ) {
            data_norm[x + y*nx] /= sqrt_sum_sqrt;
        }
    }

    step(result, data_norm, ny, nx);
}